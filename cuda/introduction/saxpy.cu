#include "hip/hip_runtime.h"
#include <stdio.h>


__global__
void saxpy(int n, float a, float*x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void) {
    int N = 1 << 20;

    float *x, *y, *x_d, *y_d;
    float *x = (float*) malloc(N * sizeof(float));
    float *y = (float*) malloc(N * sizeof(float));
    hipMallocManaged(&x_d, N * sizeof(float));
    hipMallocManaged(&y_d, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyHostToDevice));
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    saxpy<<<numBlocks, blockSize>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), cudeMemcpyDeviceToHost));


    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 4.0f));
    }
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}
