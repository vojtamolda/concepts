#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


__global__
void add(int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    const int N = 1 << 20;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    printf("Max error: %f\n", maxError);

    hipFree(x);
    hipFree(y);
}
