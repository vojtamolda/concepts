#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


__global__
void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    const int N = 1 << 20;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add<<<1, 1>>>(N, x, y);
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    printf("Max error: %f\n", maxError);

    hipFree(x);
    hipFree(y);
}
